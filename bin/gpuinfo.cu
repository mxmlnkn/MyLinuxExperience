#include "hip/hip_runtime.h"
/**
 * compile with:
 *   'nvcc' -ccbin=/usr/bin/g++-4.9 -std=c++11 --compiler-options -Wall,-Wextra -DCUDACOMMON_GPUINFO_MAIN -o gpuinfo --x cu gpuinfo.cu -lcuda && ./gpuinfo
 */

#ifndef CUDACOMMON_GPUINFO_MAIN
#   pragma once
#endif

#include <cassert>
#include <cstdint>                      // uint64_t
#include <cstdio>
#include <cstdlib>                      // NULL, malloc, free, memset
#include <cstdlib>                      // EXIT_FAILURE, exit
#include <iostream>
#include <stdexcept>
#include <sstream>
#ifdef __HIPCC__
#   include <hip/hip_runtime_api.h>
#   include <hip/hip_runtime.h>                    // hipDeviceGetAttribute
#endif


#define __FILENAME__ (__builtin_strrchr(__FILE__, '/') ? __builtin_strrchr(__FILE__, '/') + 1 : __FILE__)

/* https://stackoverflow.com/questions/8796369/cuda-and-nvcc-using-the-preprocessor-to-choose-between-float-or-double
It seems you might be conflating two things - how to differentiate between the host and device compilation trajectories when nvcc is processing CUDA code, and how to differentiate between CUDA and non-CUDA code. There is a subtle difference between the two. __CUDA_ARCH__ answers the first question, and __HIPCC__ answers the second.
*/
#if defined( __HIPCC__ )

inline void checkCudaError
(
    hipError_t  const rValue,
    char const * const file,
    int          const line
)
{
    if ( rValue != hipSuccess )
    {
        std::cout << "CUDA error " << (int) rValue << " in " << file
                  << " line:" << line << " : "
                  << hipGetErrorString( rValue ) << "\n";
        exit( EXIT_FAILURE );
    }
}
#define CUDA_ERROR(X) checkCudaError( X, __FILENAME__, __LINE__ );

#endif

/* make this header work even when not using CUDA */
#if ! defined( __HIPCC__ ) && ! defined( __host__ ) && ! defined( __device__ )
#   define __host__
#   define __device__
#endif


/**
 * some helper function to be used in templated kernels to e.g. get the
 * corresponing CUDA vector 4 for a given template parameter
 * http://www.icl.utk.edu/~mgates3/docs/cuda.html
 */
template< typename T > struct CudaVec3;
template< typename T > struct CudaVec4;
template< typename T > struct CudaVec3To4;
template< typename T > struct CudaVec4To3;
#define TMP_CUDAVECS( ELEMENTTYPE, CUDATYPENAME ) \
template<> struct CudaVec3< ELEMENTTYPE >{ typedef CUDATYPENAME##3 value_type; }; \
template<> struct CudaVec4< ELEMENTTYPE >{ typedef CUDATYPENAME##4 value_type; }; \
template<> struct CudaVec3To4< CUDATYPENAME##3 >{ typedef CUDATYPENAME##4 value_type; }; \
template<> struct CudaVec4To3< CUDATYPENAME##4 >{ typedef CUDATYPENAME##3 value_type; };
#define TMP_CUDAVECS_UI( ELEMENTTYPE, CUDATYPENAME ) \
TMP_CUDAVECS( u##ELEMENTTYPE, u##CUDATYPENAME ) \
TMP_CUDAVECS( ELEMENTTYPE, CUDATYPENAME )
TMP_CUDAVECS_UI( int8_t , char  )
TMP_CUDAVECS_UI( int16_t, short )
TMP_CUDAVECS_UI( int32_t, int   )
TMP_CUDAVECS_UI( int64_t, long  )
#undef TMP_CUDAVECS
#undef TMP_CUDAVECS_UI

/**
 * Some debug output for understanding compilation
 * @see http://docs.nvidia.com/cuda/cuda-compiler-driver-nvcc/index.html#cuda-arch
 * Sources get compiled two times __CUDA_ARCH__
 * @see https://devtalk.nvidia.com/default/topic/516937/__cuda_arch__-undefined-33-/
 * @see http://www.mersenneforum.org/showthread.php?t=18668
 * @see https://devtalk.nvidia.com/default/topic/496061/is-__cuda_arch__-broken-/
 * Normally __CUDA_ARCH__ shouldn't be used in headers! If so, watch out, that
 * is only used inside the function body, so that for host compilation it still
 * is visible... then again if it is a device function, why does it have to
 * be visible ... I'm confused
 * @see http://docs.nvidia.com/cuda/cuda-compiler-driver-nvcc/index.html#cuda-compilation-trajectory
 * @see http://docs.nvidia.com/cuda/cuda-compiler-driver-nvcc/index.html#using-separate-compilation-in-cuda
 * @see https://github.com/pathscale/nvidia_sdk_samples/blob/master/vectorAdd/build/cuda/5.0.35-13978363_x64/include/sm_30_intrinsics.h
 *   => they use #if ! defined( __CUDA_ARCH__ ) || __CUDA_ARCH__ >= 300
 *      for body-code it shouldn't matter, but for function headers to be
 *      seen this is the working approach
 */
#if 0

#if defined( __HIPCC__ )
#   warning __HIPCC__ defined
#else
#   warning __HIPCC__ NOT defined
#endif

#if defined( __CUDA_ARCH__ )
#   if __CUDA_ARCH__ < 300
#       warning __CUDA_ARCH__ < 300
#   elif __CUDA_ARCH__ <= 300
#       warning __CUDA_ARCH__ == 300
#   elif __CUDA_ARCH__ <= 350
#       warning __CUDA_ARCH__ in (300,350]
#   elif __CUDA_ARCH__ <= 400
#       warning __CUDA_ARCH__ in (350,400]
#   elif __CUDA_ARCH__ <= 500
#       warning __CUDA_ARCH__ in (400,500]
#   elif __CUDA_ARCH__ <= 600
#       warning __CUDA_ARCH__ in (500,600]
#   else
#       warning __CUDA_ARCH__ > 300
#   endif
#else
#   warning __CUDA_ARCH__ NOT defined!
#endif

#endif

/**
 * Some overloads to automatically use SIMD intrinsics if available, if not
 * then this just saves boiler-plate code, dunno why this isn't overloaded
 * like this by default at least not in CUDA 7 ...
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#simd-video
 * preliminary benchmarks indicated using __vadd2 being slower than two normal
 * adds, therefore deactivate wtih false
 */
#if defined( __CUDA_ARCH__ ) && __CUDA_ARCH__ >= 300 & false
#   define TMP_OPERATORP_UI4( UI )                                             \
    __device__ inline UI##char4 operator+                                      \
    (                                                                          \
        UI##char4 const & x,                                                   \
        UI##char4 const & y                                                    \
    )                                                                          \
    {                                                                          \
        UI##char4 z;                                                           \
        * reinterpret_cast< unsigned int * >( & z ) = __vadd4(                 \
            * reinterpret_cast< unsigned int const * >( & x ),                 \
            * reinterpret_cast< unsigned int const * >( & y )                  \
        );                                                                     \
        return z;                                                              \
    }                                                                          \
                                                                               \
    __device__ inline UI##short4 operator+                                     \
    (                                                                          \
        UI##short4 const & x,                                                  \
        UI##short4 const & y                                                   \
    )                                                                          \
    {                                                                          \
        UI##short4 z;                                                          \
        * reinterpret_cast< unsigned int * >( & z ) = __vadd2(                 \
            * reinterpret_cast< unsigned int const * >( & x ),                 \
            * reinterpret_cast< unsigned int const * >( & y )                  \
        );                                                                     \
        *( reinterpret_cast< unsigned int * >( & z ) + 1 ) = __vadd2(          \
            *( reinterpret_cast< unsigned int const * >( & x ) + 1 ),          \
            *( reinterpret_cast< unsigned int const * >( & y ) + 1 )           \
        );                                                                     \
        return z;                                                              \
    }
    TMP_OPERATORP_UI4()
    TMP_OPERATORP_UI4(u)
    #undef TMP_OPERATORP_UI4
#else
    /**
     * without explicit conversions, yields narrowing warning, because of this:
     * https://stackoverflow.com/questions/4814668/addition-of-two-chars-produces-int
     * ... whyyyyy, I'm dying a bit inside not to talk about the time I lost
     * tracking this down
     */
    __device__ inline char3 operator+( char3 const & x, char3 const & y ) {
        return { char(x.x + y.x), char(x.y + y.y), char(x.z + y.z) }; }
    __device__ inline short3 operator+( short3 const & x, short3 const & y ) {
        return { short(x.x + y.x), short(x.y + y.y), short(x.z + y.z) }; }
    __device__ inline uchar3 operator+( uchar3 const & x, uchar3 const & y )
    {
        return { (unsigned char)(x.x + y.x),
                 (unsigned char)(x.y + y.y),
                 (unsigned char)(x.z + y.z) };
     }
    __device__ inline ushort3 operator+( ushort3 const & x, ushort3 const & y )
    {
        return { (unsigned short)(x.x + y.x),
                 (unsigned short)(x.y + y.y),
                 (unsigned short)(x.z + y.z) };
    }

    __device__ inline char4 operator+( char4 const & x, char4 const & y ) {
        return { char(x.x + y.x), char(x.y + y.y), char(x.z + y.z), char(x.w + y.w) }; }
    __device__ inline short4 operator+( short4 const & x, short4 const & y ) {
        return { short(x.x + y.x), short(x.y + y.y), short(x.z + y.z), short(x.w + y.w) }; }
    __device__ inline uchar4 operator+( uchar4 const & x, uchar4 const & y )
    {
        return { (unsigned char)(x.x + y.x),
                 (unsigned char)(x.y + y.y),
                 (unsigned char)(x.z + y.z),
                 (unsigned char)(x.w + y.w) };
     }
    __device__ inline ushort4 operator+( ushort4 const & x, ushort4 const & y )
    {
        return { (unsigned short)(x.x + y.x),
                 (unsigned short)(x.y + y.y),
                 (unsigned short)(x.z + y.z),
                 (unsigned short)(x.w + y.w) };
    }
#endif
__device__ inline int3 operator+( int3 const & x, int3 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z }; }
__device__ inline long3 operator+( long3 const & x, long3 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z }; }
__device__ inline uint3 operator+( uint3 const & x, uint3 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z }; }
__device__ inline ulong3 operator+( ulong3 const & x, ulong3 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z }; }

__device__ inline int4 operator+( int4 const & x, int4 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z, x.w + y.w }; }
__device__ inline long4 operator+( long4 const & x, long4 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z, x.w + y.w }; }
__device__ inline uint4 operator+( uint4 const & x, uint4 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z, x.w + y.w }; }
__device__ inline ulong4 operator+( ulong4 const & x, ulong4 const & y ) {
    return { x.x + y.x, x.y + y.y, x.z + y.z, x.w + y.w }; }

/**
 * It is utterly confusing that
 * #if defined( __CUDA_ARCH__ ) && __CUDA_ARCH__ >= 0
 *   __device__ inline int f( void ){}
 * #elif defined( __CUDA_ARCH__ )
 *   __device__ inline int f( void ){}
 * #endif
 * does not work if called from inside a __global__ function, but the
 * lower code still seems to never be used ...
 */


#ifdef __HIPCC__
#if ! defined( __CUDA_ARCH__ ) || __CUDA_ARCH__ >= 300

/**
 * Reduces a value inside each warp who calls this function recursively
 * and the thread with lane ID 0 will have the result. Actually all threads
 * in the warps hould have the same result, as there is no predicate being
 * used for the addition and even though the shuffling rotates, it is
 * symmetrical to shifts, therefore it must have the same result for each
 * thread.
 * @verbatim
 * |0|1|2|3|4|5|6|7|
 * +-+-+-+-+-+-+-+-+
 *         .'.'.'.'  delta = 4, i.e. 0 gets the value
 *       .'.'.'.'    of lane 4, 1 of lane 5, ...
 *     .'.'.'.'
 *   .'.'.'.'
 * |0|1|2|3|4|5|6|7|
 * +-+-+-+-+-+-+-+-+
 *     .'.'          delta = 2
 *   .'.'
 * |0|1|2|3|4|5|6|7|
 * +-+-+-+-+-+-+-+-+
 *   .'              delta = 1
 * |0|1|2|3|4|5|6|7|
 * +-+-+-+-+-+-+-+-+
 * @endverbatim
 * @see https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
 */
template< typename T > __inline__ __device__
T warpReduceSum( T x )
{
#if 0
    #pragma unroll
    for ( int delta = warpSize >> 1; delta > 0; delta >>= 1 )
        x += __shfl_down( x, delta );
#else
    assert( warpSize == 32 );
    x += __shfl_down( x, 16 );
    x += __shfl_down( x,  8 );
    x += __shfl_down( x,  4 );
    x += __shfl_down( x,  2 );
    x += __shfl_down( x,  1 );
#endif
    return x;
}

template< typename T > __inline__ __device__
T warpAllReduceSum( T x )
{
#if 0
    #pragma unroll
    for ( int mask = warpSize >> 1; mask > 0; mask >>= 1 )
        x += __shfl_xor( x, mask );
#else
    assert( warpSize == 32 );
    x += __shfl_xor( x, 16 );
    x += __shfl_xor( x,  8 );
    x += __shfl_xor( x,  4 );
    x += __shfl_xor( x,  2 );
    x += __shfl_xor( x,  1 );
#endif
    return x;
}

/**
 * Similar to warpReduceSum, but actually returns the cumulative sum
 * of all elements with lower threadIds, such that it can be used to
 * calculate the cumulative sums inside a kernel which filters by linear
 * thread ID.
 * It is down recursively with a divide and conquer scheme, i.e. at each
 * step each thread only has the cumsum in as viewn in an interval of 2,4,8,...
 * E.g. if each thread would store a 1, being denoted as a block, then
 * the result of each thread having the corresponding cumsum, would look
 * like a triangle and the intermediary steps would looke like the following:
 * @verbatim
 * 5 4 3 2 1 0 9 8 7 6 5 4 3 2 1 0
 *
 * x x x x x x x x x x x x x x x x
 *
 *               |
 *               v
 *
 * x x x x x x x x x x x x x x x x
 * x   x   x   x   x   x   x   x
 *
 *               |
 *               v
 *
 * ,-+-+-. ,-+-+-. ,-+-+-.
 * v v v | v v v | v v v |
 *       |       |       |
 * x x x x x x x x x x x x x x x x
 * x x x   x x x   x x x   x x x
 * x x     x x     x x     x x
 * x       x       x       x
 *
 *               |
 *               v
 *
 * ,-+-+-+-+-+-+-+-.
 * v v v v v v v v |
 *                 |
 * x x x x x x x x x x x x x x x x
 * x x x x x x x   x x x x x x x
 * x x x x x x     x x x x x x
 * x x x x x       x x x x x
 * x x x x         x x x x
 * x x x           x x x
 * x x             x x
 * x               x
 * @endverbatim
 * Would be a bitch to do with __shfl_down, instead use __shfl_idx
 * Basically we just need to mask out some bits of the src laneId,
 * to get the target! The higher bits are basically indexing the
 * subintervals and the lower bits those inside the subintervals
 *
 * @see https://devblogs.nvidia.com/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
 *   -> since CUDA 9 they got renamed to have a suffix '_sync'
 * Actually we could delegate some of the bitmasking to CUDA by using the
 * width parameter!
 */
template< typename T > __device__ inline
T warpReduceCumSum( T x )
{
    /* todo */
    assert( warpSize == 32 );
    /* first calculate y_i = \sum_{k=0}^{2^i}  x_i */
    /* & (32-1) -> laneID, not sure if faster than % warpSize */;
    int const laneId = threadIdx.x & 0x1F;
#if 0
    for ( int width = 1; width < warpSize; width <<= 1 )
    {
        int const srcId = ( laneId & ~( width-1 ) ) - 1;
        int const dx = __shfl( x, srcId );
        if ( laneId % ( width * 2 ) >= width )
            x += dx;
    }
#else
    T dx;
    assert( warpSize == 32 );
    /**
     * using that l % 2^k is same as l &~( 2^k-1 ) and that
     * x & 0b0001111 >= 0b0001000 would be the same as checking
     * whether bit 4 was set, i.e. x & 0b0001000 != 0
     * lastly use that __shfl( x, laneId & 0b111100 - 1 )
     * would be the same as as: __shfl( x, id = 0b11, width = 0b1000 )
     */
    #if 0
        dx = __shfl( x, ( laneId & 0xFFFF ) - 1 ); if ( laneId %  2 >=  1 ) x += dx;
        dx = __shfl( x, ( laneId & 0xFFFE ) - 1 ); if ( laneId %  4 >=  2 ) x += dx;
        dx = __shfl( x, ( laneId & 0xFFFC ) - 1 ); if ( laneId %  8 >=  4 ) x += dx;
        dx = __shfl( x, ( laneId & 0xFFF8 ) - 1 ); if ( laneId % 16 >=  8 ) x += dx;
        dx = __shfl( x, ( laneId & 0xFFF0 ) - 1 ); if ( laneId % 32 >= 16 ) x += dx;
    #else
        dx = __shfl( x,  0,  2 ); if ( laneId &  1 ) x += dx;
        dx = __shfl( x,  1,  4 ); if ( laneId &  2 ) x += dx;
        dx = __shfl( x,  3,  8 ); if ( laneId &  4 ) x += dx;
        dx = __shfl( x,  7, 16 ); if ( laneId &  8 ) x += dx;
        dx = __shfl( x, 15, 32 ); if ( laneId & 16 ) x += dx;
    #endif
#endif
    return x;
}

/**
 * same as warpReduceCumSum, but only can input bool, not a 32-bit number
 * __popc returns int @see http://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__INT.html#group__CUDA__MATH__INTRINSIC__INT_1g43c9c7d2b9ebf202ff1ef5769989be46
 */
__device__ inline int warpReduceCumSumPredicate( bool const x )
{
    /* __ballot deprecated since CUDA 9 sets laneId-th bit set, i.e. lower are
     * to the "right" i.e. wil result in lower numbers */
    assert( warpSize == 32 );
    int const laneId = threadIdx.x & 0x1F; /* 32-1 -> laneID, not sure if faster than % warpSize */;
    int const mask = ( 2u << laneId ) - 1u; // will even wark for laneId 31, reslting in 0-1=-1
    return __popc( __ballot(x) & mask );
}

__device__ inline int warpReduceSumPredicate( bool const x )
{
    /* Inactive threads are represented by 0 bit!
     * @see https://stackoverflow.com/questions/23589734/ballot-behavior-on-inactive-lanes?rq=1 */
    return __popc( __ballot(x) );
}

/**
 * return type is int, because that's the return type for popc
 * and therefore by extension the one for __shfl, therefore typecasts
 * would be avoidable if you need something else
 *
 * Giving a __shared__ memory pointer like this only works for
 * __CUDA_ARCH__ >= 200
 */
__device__ inline int blockReduceCumSumPredicate( bool const x, int * const smBuffer )
{
    assert( threadIdx.y == 0 );
    assert( threadIdx.z == 0 );
    assert( blockDim.x <= warpSize * warpSize );
    /* calculate cum sums per warp */
    int cumsum = warpReduceCumSumPredicate( x );
    /* write all largest cumSums per warp, i.e. highest threadIdx / laneId
     * into __shared__ buffer. The highest thread doesn't need to store
     * its value, because noone needs to add it, this is useful, because
     * that allows calling this with some higher threadIds being filtered out */
    int const iSubarray = threadIdx.x / warpSize;
    if ( threadIdx.x % warpSize == warpSize - 1 )
        smBuffer[ iSubarray ] = cumsum;
    /* the first warp now reduces these intermediary sums to another cumsum
     * and writes it back. This is enough assuming that warpSize * warpSize <=
     * maxThreadsPerBlock, which is the case normally, i.e. 32^2 <= 1024 */
    /* DOOOOOOOOOOOOOOOOOOOOOOOOONNNNNNNTTTTTTTTT put __syncthreads inside
     * an if-statement if you ever wanna use it ouside the if statement again!
     * The bug caused by using this commented code, cost me hours to track
     * down ... */
    /*
    if ( threadIdx.x < warpSize )
    {
        __syncthreads();
        int const globalCumSum = warpReduceCumSum( smBuffer[ threadIdx.x ] );
        __syncthreads();
        smBuffer[ threadIdx.x ] = globalCumSum;
    }
    */
    __syncthreads();
    int globalCumSum;
    if ( threadIdx.x < warpSize )
        globalCumSum = warpReduceCumSum( smBuffer[ threadIdx.x ] );
    __syncthreads();
    if ( threadIdx.x < warpSize )
        smBuffer[ threadIdx.x ] = globalCumSum;
    /* now we need to apply these cumsums which kinda act like global offsets
     * to all our "small" cumsum variations */
    __syncthreads();
    if ( iSubarray > 0 )
        cumsum += smBuffer[ iSubarray-1 ];
    /* wait until everyone read before giving control back which could
     * possibly mess up the buffer! */
    __syncthreads();
    return cumsum;
}

/**
 * exactly same as blockReduceCumSumPredicate, but loose the 'Predicate'
 * suffix in al function names
 */
__device__ inline int blockReduceCumSum( int const x, int * const smBuffer )
{
    assert( threadIdx.y == 0 );
    assert( threadIdx.z == 0 );
    assert( blockDim.x <= warpSize * warpSize );
    int cumsum = warpReduceCumSum( x );
    int const iSubarray = threadIdx.x / warpSize;
    if ( threadIdx.x % warpSize == warpSize - 1 )
        smBuffer[ iSubarray ] = cumsum;
    __syncthreads();
    int globalCumSum;
    if ( threadIdx.x < warpSize )
        globalCumSum = warpReduceCumSum( smBuffer[ threadIdx.x ] );
    __syncthreads();
    if ( threadIdx.x < warpSize )
        smBuffer[ threadIdx.x ] = globalCumSum;
    __syncthreads();
    if ( iSubarray > 0 )
        cumsum += smBuffer[ iSubarray-1 ];
    __syncthreads();
    return cumsum;
}

__device__ inline int blockReduceSumPredicate( bool const x, int * const smBuffer )
{
    assert( threadIdx.y == 0 );
    assert( threadIdx.z == 0 );
    assert( blockDim.x <= warpSize * warpSize );
    int sum = warpReduceSumPredicate( x );
    if ( threadIdx.x < warpSize )
        smBuffer[ threadIdx.x ] = 0;
    __syncthreads();
    if ( threadIdx.x % warpSize == 0 )
        smBuffer[ threadIdx.x / warpSize ] = sum;
    __syncthreads();
    if ( threadIdx.x < warpSize )
        sum = warpReduceSum( smBuffer[ threadIdx.x ] );
    __syncthreads();
    // only threadIdx.x == 0 has the correct value!
    return sum;
}

/**
 * same as blockReduceSumPredicate just delate the "Predicate" inside first
 * warp reduce function name
 */
__device__ inline int blockReduceSum( int const x, int * const smBuffer )
{
    assert( threadIdx.y == 0 );
    assert( threadIdx.z == 0 );
    assert( blockDim.x <= warpSize * warpSize );
    int sum = warpReduceSum( x );
    if ( threadIdx.x < warpSize )
        smBuffer[ threadIdx.x ] = 0;
    __syncthreads();
    int const iSubarray = threadIdx.x / warpSize;
    if ( threadIdx.x % warpSize == 0 )
        smBuffer[ iSubarray ] = sum;
    __syncthreads();
    if ( threadIdx.x < warpSize )
        sum = warpReduceSum( smBuffer[ threadIdx.x ] );
    // only threadIdx.x == 0 has the correct value!
    return sum;
}
#endif // __CUDA_ARCH__ >= 300

#endif // __HIPCC__


template< typename T, typename S >
__host__ __device__
inline T ceilDiv( T a, S b )
{
    assert( b != S(0) );
    assert( a == a );
    assert( b == b );
    return ( a + b - T(1) ) / b;
}

#include <sstream>
#include <string>
#include <vector>

/**
 * Given the number of bytes, this function prints out an exact human
 * readable format, e.g. 128427:
 *   logical: 125 MiB 427 B
 *   SI     : 128 MB 427 B
 * Wasn't intented, but in both representations the amount of bytes are
 * identical for this number. (This works if xMiB * 1024 ends on 000)
 */
inline std::string prettyPrintBytes
(
    size_t       bytes,
    bool   const logical = true
)
{
    char const suffixes[] = { ' ', 'k', 'M', 'G', 'T', 'P', 'E', 'Z', 'Y' };
    std::stringstream out;
    std::vector< size_t > parts;
    for ( unsigned i = 0u; i < sizeof( suffixes ); ++i )
    {
        parts.push_back( bytes % size_t( 1024 ) );
        bytes /= size_t( 1024 );
        if ( bytes == 0 )
            break;
    }
    assert( parts.size() > 0 );
    for ( int i = (int) parts.size()-1; i >= 0; --i )
    {
        if ( i != (int) parts.size()-1 && parts.at(i) == 0 )
            continue;
        out << parts[i] << " " << suffixes[i] << ( logical ? "i" : "" )
            << "B" << ( i > 0 ? " " : "" );
    }
    std::string result = out.str();
    result.erase( result.size()-1, 1 );
    return result;
}

#if defined( __HIPCC__ )

/**
 * Chooses an optimal configuration for number of blocks and number of threads
 * Note that every kernel may have to calculate on a different amount of
 * elements, so this needs to be calculated inside the kernel with:
 *    for ( i = linid; i < nElements; i += nBlocks * nThreads )
 * which yields the following number of iterations:
 *    nIterations = (nElements-1 - linid) / ( nBlocks * nThreads ) + 1
 * derivation:
 *    search for highest n which satisfies i + n*s <= m-1
 *    note that we used <= m-1 instead of < m to work with floor later on
 *    <=> search highest n: n <= (m-1-i)/s
 *    which is n = floor[ (m-1-i)/s ]. Note that floor wouldn't be possible
 *    for < m, because it wouldn't account for the edge case for (m-1-i)/s == n
 *    the highest n means the for loop will iterate with i, i+s, i+2*s, i+...n*s
 *    => nIterations = n+1 = floor[ (m-1-i)/s ] + 1
 */
inline void calcKernelConfig( int iDevice, uint64_t n, int * nBlocks, int * nThreads )
{
    int const nMaxThreads  = 256;
    int const nMinElements = 32; /* The assumption: one kernel with nMinElements work won't be much slower than nMinElements kernels with each 1 work element. Of course this is workload / kernel dependent, so the fixed value may not be the best idea */

    /* set current device and get device infos */
    int nDevices;
    CUDA_ERROR( hipGetDeviceCount( &nDevices ) );
    assert( iDevice < nDevices );
    CUDA_ERROR( hipSetDevice( iDevice ) );

    // for GTX 760 this is 12288 threads per device and 384 real cores
    hipDeviceProp_t deviceProperties;
    CUDA_ERROR( hipGetDeviceProperties( &deviceProperties, iDevice) );

    int const nMaxThreadsGpu = deviceProperties.maxThreadsPerMultiProcessor
                             * deviceProperties.multiProcessorCount;
    if ( n < (uint64_t) nMaxThreadsGpu * nMinElements )
    {
        uint64_t const nThreadsNeeded = ceilDiv( n, nMinElements );
        *nBlocks  = ceilDiv( nThreadsNeeded, nMaxThreads );
        *nThreads = nMaxThreads;
        if ( *nBlocks == 1 )
        {
            assert( nThreadsNeeded <= nMaxThreads );
            *nThreads = nThreadsNeeded;
        }
    }
    else
    {
        *nBlocks  = nMaxThreadsGpu / nMaxThreads;
        *nThreads = nMaxThreads;
    }
    assert( *nBlocks > 0 );
    assert( *nThreads > 0 );
    uint64_t nIterations = 0;
    for ( uint64_t linid = 0; linid < (uint64_t) *nBlocks * *nThreads; ++linid )
    {
        /* note that this only works if linid < n */
        assert( linid < n );
        nIterations += (n-linid-1) / ( *nBlocks * *nThreads ) + 1;
        //printf( "[thread %i] %i elements\n", linid, (n-linid) / ( *nBlocks * *nThreads ) );
    }
    //printf( "Total %i elements out of %i wanted\n", nIterations, n );
    assert( nIterations == n );
}


inline __device__ unsigned long long int getLinearThreadId( void )
{
    unsigned long long int i    = threadIdx.x;
    unsigned long long int iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax;
    // expands to: i = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x

    return i;
}

inline __device__ unsigned long long int getLinearGlobalId( void )
{
    unsigned long long int i    = threadIdx.x;
    unsigned long long int iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax; iMax *= blockDim.z;
    i +=  blockIdx.x * iMax; iMax *= gridDim.x;
    i +=  blockIdx.y * iMax; iMax *= gridDim.y;
    i +=  blockIdx.z * iMax;

    return i;
}

#include <utility>                  // pair

inline __device__ void getLinearGlobalIdSize
(
    unsigned long long int * riThread,
    unsigned long long int * rnThreads
)
{
    unsigned long long int & i    = *riThread ;
    unsigned long long int & iMax = *rnThreads;

    i    = threadIdx.x;
    iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax; iMax *= blockDim.z;
    i +=  blockIdx.x * iMax; iMax *= gridDim.x;
    i +=  blockIdx.y * iMax; iMax *= gridDim.y;
    i +=  blockIdx.z * iMax; iMax *= gridDim.z;
}

inline __device__ unsigned long long int getLinearBlockId( void )
{
    unsigned long long int i    = blockIdx.x;
    unsigned long long int iMax = gridDim.x;

    i += blockIdx.y * iMax; iMax *= gridDim.y;
    i += blockIdx.z * iMax;
    // expands to: i = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x

    return i;
}

inline __device__ unsigned long long int getBlockSize( void )
{
    return blockDim.x * blockDim.y * blockDim.z;
}

inline __device__ unsigned long long int getGridSize( void )
{
    return gridDim.x * gridDim.y * gridDim.z;
}

#endif // __HIPCC__


#include <cassert>
#include <cstdio>               // printf, fflush
#include <cstdlib>              // NULL, malloc, free
#include <map>


/**
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
 * @see https://devtalk.nvidia.com/default/topic/763273/peak-performance-of-integer-operation/
 * "Multiple instructions" means there is no native instruction to perform that operation, and instead the compiler emits an instruction sequence to perform the operation. It is typically on the order of 5-50 instructions. This can vary from operation to operation, architecture to architecture, and even compiler version to compiler version. If you want to find out what it is for a specific case, create a small test code, compile it, and then dump the machine code using
 *   cuobjdump -sass mycode
 * 32-bit integer add is at approximately the same throughput as corresponding floating point operations for all architectures. So I guess your concern is primarily around the 32-bit integer multiply.
 * => signaled here in the table with -1
 * @see https://devtalk.nvidia.com/default/topic/948014/forward-looking-gpu-integer-performance/?offset=14
 */
static std::map< std::string, std::vector< int > > sCudaInstructionThroughput =
{
                 /* 3.0 & 3.2, 3.5 & 3.7, 5.0 & 5.2, 5.3, 6.0, 6.1, 6.2, 7.0 */
    { "hpfpfma"    , {   0,   0,   0, 256, 128,   2, 256, 128 } },
    { "spfpfma"    , { 192, 192, 128, 128,  64, 128, 128,  64 } },
    { "dpfpfma"    , {   8,  64,   4,   4,  32,   4,   4,  32 } },
    { "sprecip"    , {  32,  32,  32,  32,  16,  32,  32,  16 } }, /* identical to special function units */
    { "32biadd"    , { 160, 160, 128, 128,  64, 128, 128,  64 } }, /* identical to spfma, except for kepler! */
    { "32bimul"    , {  32,  32,  -1,  -1,  -1,  -1,  -1,  64 } },
    { "32bishift"  , {  32,  64,  64,  64,  32,  64,  64,  64 } }, /* 2x special function, except 3.0, 3.2 */
    { "cmp"        , { 160, 160,  64,  64,  32,  64,  64,  64 } },
    { "32bireverse", {  32,  32,  64,  64,  32,  64,  64,  -1 } },
    { "32biand"    , { 160, 160, 128, 128,  64, 128, 128,  64 } }, /* identical to 32biadd */
    { "popc"       , {  32,  32,  32,  32,  16,  32,  32,  16 } }, /* identical to sprecip */
    { "shfl"       , {  32,  32,  32,  32,  32,  32,  32,  32 } }, /* always: 32 */
    { "convto32bit", { 128, 128,  32,  32,  16,  32,  32,  16 } },
    { "conv64bit"  , {   8,  32,   4,   4,  16,   4,   4,  16 } },
    { "convmisc"   , {  32,  32,  32,  32,  16,  32,  32,  16 } }  /* identical to sprecip */
};

/**
 * Returns the number of arithmetic CUDA cores per streaming multiprocessor
 * Note that there are also extra special function units.
 * This corresponds to 32 bit add, multiply, FMA instructions.
 * 64-bit capabilities will be (far) less
 * Note that for 2.0 the two warp schedulers can only issue 16 instructions
 * per cycle each. Meaning the 32 CUDA cores can't be used in parallel with
 * the 4 special function units. For 2.1 up this is a different matter
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
 *      from CUDA Toolkit v9.0.176
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions
 **/
inline int getCudaCoresPerMultiprocessor
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 2 && minorVersion == 0 ) /* Fermi   */ return 32 ;
    if ( majorVersion == 2 && minorVersion == 1 ) /* Fermi   */ return 48 ;
    if ( majorVersion == 3 )                      /* Kepler  */ return 192;
    if ( majorVersion == 5 )                      /* Maxwell */ return 128;
    if ( majorVersion == 6 && minorVersion == 0 ) /* Pascal  */ return 64 ;
    if ( majorVersion == 6 && minorVersion == 1 ) /* Pascal  */ return 128;
    if ( majorVersion == 6 && minorVersion == 2 ) /* Pascal  */ return 128;
    if ( majorVersion == 7 )                      /* Volta   */ return 64 ;
    return 0; /* unknown, could also throw exception */
}

inline int getSpecialFunctionUnitsPerMultiprocessor
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 3 )                      /* Kepler  */ return 32;
    if ( majorVersion == 5 )                      /* Maxwell */ return 32;
    if ( majorVersion == 6 && minorVersion == 0 ) /* Pascal  */ return 16;
    if ( majorVersion == 6 && minorVersion == 1 ) /* Pascal  */ return 32;
    if ( majorVersion == 6 && minorVersion == 2 ) /* Pascal  */ return 32;
    if ( majorVersion == 7 )                      /* Volta   */ return 16;
    return 0; /* unknown, could also throw exception */
}

inline int getWarpSchedulersPerMultiprocessor
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 3 )                      /* Kepler  */ return 4;
    if ( majorVersion == 5 )                      /* Maxwell */ return 4;
    if ( majorVersion == 6 && minorVersion == 0 ) /* Pascal  */ return 2;
    if ( majorVersion == 6 && minorVersion == 1 ) /* Pascal  */ return 4;
    if ( majorVersion == 6 && minorVersion == 2 ) /* Pascal  */ return 4;
    if ( majorVersion == 7 )                      /* Volta   */ return 4;
    return 0; /* unknown, could also throw exception */
}

inline int getDoublePrecisionUnitsPerMultiprocessor
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 3 && minorVersion == 0 ) /* Kepler  */ return 8 ;
    if ( majorVersion == 3 && minorVersion == 2 ) /* Kepler  */ return 8 ;
    if ( majorVersion == 3 && minorVersion == 5 ) /* Kepler  */ return 64; /* 8 for GeForce GPUs .. how to differentiate? */
    if ( majorVersion == 3 && minorVersion == 7 ) /* Kepler  */ return 64;
    if ( majorVersion == 5 )                      /* Maxwell */ return 4 ;
    if ( majorVersion == 6 && minorVersion == 0 ) /* Pascal  */ return 32;
    if ( majorVersion == 6 && minorVersion == 1 ) /* Pascal  */ return 4 ;
    if ( majorVersion == 6 && minorVersion == 2 ) /* Pascal  */ return 4 ;
    if ( majorVersion == 7 )                      /* Volta   */ return 32;
    return 0; /* unknown, could also throw exception */
}

/**
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
 *      from CUDA Toolkit v9.0.176
 **/
inline int getCudaMaxConcurrentKernels
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 2 ) /* Fermi */ return 0;
    if ( majorVersion == 3 && minorVersion == 0 ) /* Kepler  */ return 16 ;
    if ( majorVersion == 3 && minorVersion == 2 ) /* Kepler  */ return 4  ;
    if ( majorVersion == 3 && minorVersion == 5 ) /* Kepler  */ return 32 ;
    if ( majorVersion == 3 && minorVersion == 7 ) /* Kepler  */ return 32 ;
    if ( majorVersion == 5 && minorVersion == 0 ) /* Maxwell */ return 32 ;
    if ( majorVersion == 5 && minorVersion == 2 ) /* Maxwell */ return 32 ;
    if ( majorVersion == 5 && minorVersion == 3 ) /* Maxwell */ return 16 ;
    if ( majorVersion == 6 && minorVersion == 0 ) /* Pascal  */ return 128;
    if ( majorVersion == 6 && minorVersion == 1 ) /* Pascal  */ return 32 ;
    if ( majorVersion == 6 && minorVersion == 2 ) /* Pascal  */ return 16 ;
    if ( majorVersion == 7 && minorVersion == 0 ) /* Volta   */ return 128;
    return 0;   /* unknown, could also throw exception */
}

/**
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capability
 *      from CUDA Toolkit v9.0.176
 */
inline std::string getCudaCodeName
(
    int const majorVersion,
    int const = 0
)
{
    if ( majorVersion == 2 ) return "Fermi"  ;
    if ( majorVersion == 3 ) return "Kepler" ;
    if ( majorVersion == 5 ) return "Maxwell";
    if ( majorVersion == 6 ) return "Pascal" ;
    if ( majorVersion == 7 ) return "Volta"  ;
    return 0; /* unknown, could also throw exception */
}


#if defined( __HIPCC__ )

/**
 * @return flops (not GFlops, ... )
 * @see https://www.techpowerup.com/gpudb/1857/geforce-gtx-760
 * As can be seen here:
 * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions
 * The throughput since at least Compute Capability 3.0 offers as many FMAs
 * as simple additions, therefore for the flops we need a factor 2 to get
 * the theoretical flops
 */
inline float getCudaPeakSPFlops( hipDeviceProp_t const & props )
{
    return (float) props.multiProcessorCount * props.clockRate /* kHz */ * 1e3f *
        getCudaCoresPerMultiprocessor( props.major, props.minor ) * 2 /* FMA */;
}

inline float getCudaPeakDPFlops( hipDeviceProp_t const & props )
{
    return (float) props.multiProcessorCount * props.clockRate /* kHz */ * 1e3f *
        getDoublePrecisionUnitsPerMultiprocessor( props.major, props.minor ) * 2 /* FMA */;
}


#include <sstream>

inline std::string getCudaCacheConfigString( void )
{
    std::stringstream out;
    out << "Prefer ";
    hipFuncCache_t funcCache;
    hipDeviceGetCacheConfig( &funcCache );
    switch ( funcCache )
    {
        case hipFuncCachePreferNone  : out << "None"  ; break;
        case hipFuncCachePreferShared: out << "Shared"; break;
        case hipFuncCachePreferL1    : out << "L1"    ; break;
        case hipFuncCachePreferEqual : out << "Equal" ; break;
        default: printf( "?" );
    }
    return out.str();
}

inline std::string getCudaSharedMemBankSizeString( void )
{
    std::stringstream out;
    hipSharedMemConfig config;
    hipDeviceGetSharedMemConfig( &config );
    switch ( config )
    {
        case hipSharedMemBankSizeDefault  : out << "Default"; break;
        case hipSharedMemBankSizeFourByte : out << "4 Bytes"; break;
        case hipSharedMemBankSizeEightByte: out << "8 Bytes"; break;
        default: printf( "?" );
    }
    return out.str();
}


inline std::string printSharedMemoryConfig( void )
{
    std::stringstream out;
    out << "[Shared Memory] Config: " << getCudaCacheConfigString()
        << ", Bank Size: " << getCudaSharedMemBankSizeString() << "\n";
    return out.str();
}

/**
 * @param[out] rpDeviceProperties - Array of hipDeviceProp_t of length rnDevices
 *             the user will need to free (C-style) this data on program exit!
 * @param[out] rnDevices - will hold number of cuda devices
 *
 * @see https://www.cs.cmu.edu/afs/cs/academic/class/15668-s11/www/cuda-doc/html/structcudaDeviceProp.html
 * Most of these can also be queried using hipDeviceGetAttribute from the
 * hip/hip_runtime_api.h header
 * @see http://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__DEVICE.html
 *      v9.0.176
 */
inline void getCudaDeviceProperties
(
    hipDeviceProp_t **       rpDeviceProperties = NULL,
    int             *       rnDevices          = NULL,
    bool              const rPrintInfo         = true
)
{
    printf( "Getting Device Informations. As this is the first command, "
            "it can take ca.30s, because the GPU must be initialized.\n" );
    fflush( stdout );

    int fallbackNDevices;
    if ( rnDevices == NULL )
        rnDevices = &fallbackNDevices;
    CUDA_ERROR( hipGetDeviceCount( rnDevices ) );

    hipDeviceProp_t * fallbackPropArray;
    if ( rpDeviceProperties == NULL )
        rpDeviceProperties = &fallbackPropArray;
    *rpDeviceProperties = (hipDeviceProp_t*) malloc( (*rnDevices) * sizeof(hipDeviceProp_t) );
    assert( *rpDeviceProperties != NULL );

    for ( int iDevice = 0; iDevice < (*rnDevices); ++iDevice )
    {
        hipDeviceProp_t * prop = &( (*rpDeviceProperties)[iDevice] );
        CUDA_ERROR( hipGetDeviceProperties( prop, iDevice ) );

        if ( not rPrintInfo )
            continue;

        if ( iDevice == 0 && prop->major == 9999 && prop->minor == 9999 )
            printf("There is no device supporting CUDA.\n");

        const char cms[5][20] =
            { "Default", "Exclusive", "Prohibited", "ExclusiveProcess", "Unknown" };
        const char * computeModeString;
        switch ( prop->computeMode )
        {
            case hipComputeModeDefault          : computeModeString = cms[0];
            case hipComputeModeExclusive        : computeModeString = cms[1];
            case hipComputeModeProhibited       : computeModeString = cms[2];
            case hipComputeModeExclusiveProcess : computeModeString = cms[3];
            default                              : computeModeString = cms[4];
        }
        int const coresPerSM = getCudaCoresPerMultiprocessor( prop->major, prop->minor );

        /**
         * @see http://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__TYPES.html#group__CUDA__TYPES_1ge12b8a782bebe21b1ac0091bf9f4e2a3
         * List of attributes not included in device properties:
         */
        #define TMP_ATTRIBUTE( VARNAME, NUMBER ) \
        int VARNAME = -1;                        \
        if ( NUMBER < CU_DEVICE_ATTRIBUTE_MAX )  \
            hipDeviceGetAttribute( &VARNAME, (hipDeviceAttribute_t) NUMBER, iDevice );
        /*
        hipDeviceAttributeMaxPitch (could be normal mem pitch?)
        hipDeviceAttributeUnifiedAddressing = 41
            Device shares a unified address space with the host
        hipDeviceAttributeMaxTexture1DLayered = 42
            Maximum 1D layered texture width
        CU_DEVICE_ATTRIBUTE_MAXIMUM_TEXTURE1D_LAYERED_LAYERS = 43
            Maximum layers in a 1D layered texture
            hipDeviceAttributeMaxTexture3DAlt = 47
            Alternate maximum 3D texture width
        hipDeviceAttributeMaxTexture3DAlt = 48
            Alternate maximum 3D texture height
        hipDeviceAttributeMaxTexture3DAlt = 49
            Alternate maximum 3D texture depth
        hipDeviceAttributeMaxTextureCubemap = 52
            Maximum cubemap texture width/height
        hipDeviceAttributeMaxTextureCubemapLayered = 53
            Maximum cubemap layered texture width/height
        CU_DEVICE_ATTRIBUTE_MAXIMUM_TEXTURECUBEMAP_LAYERED_LAYERS = 54
            Maximum layers in a cubemap layered texture
        hipDeviceAttributeMaxSurface1D = 55
            Maximum 1D surface width
        hipDeviceAttributeMaxSurface2D = 56
            Maximum 2D surface width
        hipDeviceAttributeMaxSurface2D = 57
            Maximum 2D surface height
        hipDeviceAttributeMaxSurface3D = 58
            Maximum 3D surface width
        hipDeviceAttributeMaxSurface3D = 59
            Maximum 3D surface height
        hipDeviceAttributeMaxSurface3D = 60
            Maximum 3D surface depth
        hipDeviceAttributeMaxSurface1DLayered = 61
            Maximum 1D layered surface width
        CU_DEVICE_ATTRIBUTE_MAXIMUM_SURFACE1D_LAYERED_LAYERS = 62
            Maximum layers in a 1D layered surface
        hipDeviceAttributeMaxSurface2DLayered = 63
            Maximum 2D layered surface width
        hipDeviceAttributeMaxSurface2DLayered = 64
            Maximum 2D layered surface height
        CU_DEVICE_ATTRIBUTE_MAXIMUM_SURFACE2D_LAYERED_LAYERS = 65
            Maximum layers in a 2D layered surface
        hipDeviceAttributeMaxSurfaceCubemap = 66
            Maximum cubemap surface width
        hipDeviceAttributeMaxSurfaceCubemapLayered = 67
            Maximum cubemap layered surface width
        CU_DEVICE_ATTRIBUTE_MAXIMUM_SURFACECUBEMAP_LAYERED_LAYERS = 68
            Maximum layers in a cubemap layered surface
        hipDeviceAttributeMaxTexture2DLinear = 72
            Maximum 2D linear texture pitch in bytes
        hipDeviceAttributeMaxTexture2DMipmap = 73
            Maximum mipmapped 2D texture width
        hipDeviceAttributeMaxTexture2DMipmap = 74
            Maximum mipmapped 2D texture height
        hipDeviceAttributeMaxTexture1DMipmap = 77
            Maximum mipmapped 1D texture width
        */
        TMP_ATTRIBUTE( bStreamPrioritiesSupported, 78 ) // hipDeviceAttributeStreamPrioritiesSupported
        TMP_ATTRIBUTE( bGlobalL1CacheSupported   , 79 ) // hipDeviceAttributeGlobalL1CacheSupported
        TMP_ATTRIBUTE( bLocalL1CacheSupported    , 80 ) // hipDeviceAttributeLocalL1CacheSupported
        TMP_ATTRIBUTE( nBytesMaxSMPerMP          , 81 ) // hipDeviceAttributeMaxSharedMemoryPerMultiprocessor
        // hipDeviceAttributeManagedMemory = 83 Device can allocate managed memory on this system
        TMP_ATTRIBUTE( nBytesMaxRegistersPerMP   , 82 ) // hipDeviceAttributeMaxRegistersPerMultiprocessor
        TMP_ATTRIBUTE( bMultiGpuBoard            , 84 ) // hipDeviceAttributeIsMultiGpuBoard
        TMP_ATTRIBUTE( iMultiGpuBoardId          , 85 ) // hipDeviceAttributeMultiGpuBoardGroupId
        TMP_ATTRIBUTE( ratioSPToDPFlops          , 87 ) // hipDeviceAttributeSingleToDoublePrecisionPerfRatio
        /*
        hipDeviceAttributePageableMemoryAccess = 88
            Device supports coherently accessing pageable memory without calling hipHostRegister on it
        hipDeviceAttributeConcurrentManagedAccess = 89
            Device can coherently access managed memory concurrently with the CPU
        hipDeviceAttributeComputePreemptionSupported = 90
            Device supports compute preemption.
        hipDeviceAttributeCanUseHostPointerForRegisteredMem = 91
            Device can access host registered memory at the same virtual address as the CPU
        CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS = 92
            hipStreamBatchMemOp and related APIs are supported.
        CU_DEVICE_ATTRIBUTE_CAN_USE_64_BIT_STREAM_MEM_OPS = 93
            64-bit operations are supported in hipStreamBatchMemOp and related APIs.
        hipDeviceAttributeCanUseStreamWaitValue = 94
            hipStreamWaitValueNor is supported.
        hipDeviceAttributeCooperativeLaunch = 95
            Device supports launching cooperative kernels via hipModuleLaunchCooperativeKernel
        hipDeviceAttributeCooperativeMultiDeviceLaunch = 96
            Device can participate in cooperative kernels launched via hipModuleLaunchCooperativeKernelMultiDevice
        hipDeviceAttributeSharedMemPerBlockOptin = 97
            Maximum optin shared memory per block
        CU_DEVICE_ATTRIBUTE_MAX
        */
        #undef TMP_ATTRIBUTE

        printf( "\n================== Device Number %i ==================\n",iDevice );
        printf( "| Device name              : %s\n"        , prop->name );
        printf( "| Computability            : %i.%i\n"     , prop->major,
                                                             prop->minor );
        printf( "| Code Name                : %s\n"        , getCudaCodeName( prop->major, prop->minor ).c_str() );
        printf( "| PCI Bus ID               : %i\n"        , prop->pciBusID );
        printf( "| PCI Device ID            : %i\n"        , prop->pciDeviceID );
        printf( "| PCI Domain ID            : %i\n"        , prop->pciDomainID );
        printf( "|------------------- Architecture -------------------\n" );
        printf( "| Number of SMX            : %i\n"        , prop->multiProcessorCount );
        printf( "| Max Threads per SMX      : %i\n"        , prop->maxThreadsPerMultiProcessor );
        printf( "| Max Threads per Block    : %i\n"        , prop->maxThreadsPerBlock );
        printf( "| Warp Size                : %i\n"        , prop->warpSize );
        printf( "| Warp Schedulers per MP   : %i\n"        , getWarpSchedulersPerMultiprocessor( prop->major, prop->minor ) );
        printf( "| Clock Rate               : %f GHz\n"    , prop->clockRate/1.0e6f );
        printf( "| Max Block Size           : (%i,%i,%i)\n", prop->maxThreadsDim[0],
                                                             prop->maxThreadsDim[1],
                                                             prop->maxThreadsDim[2] );
        printf( "| Max Grid Size            : (%i,%i,%i)\n", prop->maxGridSize[0],
                                                             prop->maxGridSize[1],
                                                             prop->maxGridSize[2] );
        printf( "|  => Max conc. Threads    : %i\n"        , prop->multiProcessorCount *
                                                             prop->maxThreadsPerMultiProcessor );
        printf( "|  => Warps per SMX        : %i\n"        , prop->maxThreadsPerMultiProcessor /
                                                             prop->warpSize );
        printf( "| CUDA Cores per Multiproc.: %i\n"        , coresPerSM );
        printf( "| Total CUDA Cores         : %i\n"        , prop->multiProcessorCount * coresPerSM );
        printf( "| Peak SP-FLOPS            : %f GFLOPS\n" , getCudaPeakSPFlops( *prop ) / 1e9f );
        printf( "| Peak DP-FLOPS            : %f GFLOPS\n" , getCudaPeakDPFlops( *prop ) / 1e9f );
        printf( "| Peak SP/DP-FLOPS         : %i (%i)\n"   , ratioSPToDPFlops, getCudaCoresPerMultiprocessor( prop->major, prop->minor ) / getDoublePrecisionUnitsPerMultiprocessor( prop->major, prop->minor ) );
        printf( "| Special Fun. Units per MP: %i\n"        , getSpecialFunctionUnitsPerMultiprocessor( prop->major, prop->minor ) );
        printf( "|---------------------- Memory ----------------------\n" );
        printf( "| Total Global Memory      : %lu Bytes\n" , prop->totalGlobalMem );
        printf( "| Total Constant Memory    : %lu Bytes\n" , prop->totalConstMem );
        printf( "| Shared Memory per Block  : %lu Bytes\n" , prop->sharedMemPerBlock );
        printf( "| Shared Memory per Multip.: %i Bytes\n"  , nBytesMaxSMPerMP );
        printf( "| Global L1 Cache supported: %s\n"        , bGlobalL1CacheSupported ? "true" : "false" );
        printf( "| Local  L1 Cache supported: %s\n"        , bLocalL1CacheSupported  ? "true" : "false" );
        printf( "| L2 Cache Size            : %u Bytes\n"  , prop->l2CacheSize );
        printf( "| Registers per Block      : %i\n"        , prop->regsPerBlock );
        printf( "| Registers per Multiproc. : %i\n"        , nBytesMaxRegistersPerMP );
        printf( "| Memory Bus Width         : %i Bits\n"   , prop->memoryBusWidth );
        printf( "| Memory Clock Rate        : %f GHz\n"    , prop->memoryClockRate/1.0e6f );
        printf( "| Memory Pitch             : %lu\n"       , prop->memPitch );
        printf( "| Unified Addressing       : %i\n"        , prop->unifiedAddressing );
        printf( "| Texture Alignment        :  %ld\n"      , prop->textureAlignment );
        printf( "| Max 1D Texture Size      : %i\n"        , prop->maxTexture1D );
        printf( "| Max 2D Texture Size      : (%i,%i)\n"   , prop->maxTexture2D[0],
                                                             prop->maxTexture2D[1] );
        // this really is ONLY in CUDA 3.2. (maybe 3.x) available Oo
        //printf( "| Max 2D Texture Array Size: (%i,%i)\n"   , prop->maxTexture2DArray[0],
        //                                                     prop->maxTexture2DArray[1] );
        printf( "| Max 3D Texture Size      : (%i,%i,%i)\n", prop->maxTexture3D[0],
                                                             prop->maxTexture3D[1] ,
                                                             prop->maxTexture3D[2] );
        printf( "| Cache Configuration      : %s\n"        , getCudaCacheConfigString().c_str() );
        printf( "| Shared Memory Bank Size  : %s\n"        , getCudaSharedMemBankSizeString().c_str() );
        printf( "|--------------------- Graphics ---------------------\n" );
        printf( "| Compute mode             : %s\n"        ,      computeModeString );
        printf( "|---------------------- Other -----------------------\n" );
        printf( "| Can map Host Memory      : %s\n"        , prop->canMapHostMemory  ? "true" : "false" );
        printf( "| Can run Kernels conc.    : %s\n"        , prop->concurrentKernels ? "true" : "false" );
        printf( "|   => max. conc. kernels  : %i\n"        , getCudaMaxConcurrentKernels( prop->major, prop->minor ) );
        printf( "| Number of Asyn. Engines  : %i\n"        , prop->asyncEngineCount );
        printf( "| Can Copy and Kernel conc.: %s\n"        , prop->deviceOverlap     ? "true" : "false" );
        printf( "| ECC Enabled              : %s\n"        , prop->ECCEnabled        ? "true" : "false" );
        printf( "| Device is Integrated     : %s\n"        , prop->integrated        ? "true" : "false" );
        printf( "| Kernel Timeout Enabled   : %s\n"        , prop->kernelExecTimeoutEnabled ? "true" : "false" );
        printf( "| Uses TESLA Driver        : %s\n"        , prop->tccDriver         ? "true" : "false" );
        printf( "| Stream Priorities Supp.  : %s\n"        , bStreamPrioritiesSupported ? "true" : "false" );
        printf( "| Multi-GPU Board          : %s\n"        , bMultiGpuBoard          ? "true" : "false" );
        if ( bMultiGpuBoard )
        printf( "| Multi-GPU Board ID       : %i\n"        , iMultiGpuBoardId );
        printf( "=====================================================\n" );
        fflush( stdout );
    }

    if ( rpDeviceProperties == &fallbackPropArray )
        free( fallbackPropArray );
}

#if defined( __CUDA_ARCH__ ) && __CUDA_ARCH__ < 600
/**
 * atomicAdd for double is not natively implemented, because it's not
 * supported by (all) the hardware, therefore resulting in a time penalty.
 * http://stackoverflow.com/questions/12626096/why-has-atomicadd-not-been-implemented-for-doubles
 * https://stackoverflow.com/questions/37566987/cuda-atomicadd-for-doubles-definition-error
 */
inline __device__
double atomicAdd( double * address, double val )
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int *) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
              __double_as_longlong( val + __longlong_as_double(assumed) ));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

#endif // __HIPCC__


template< class T >
class MirroredVector;

template< class T >
class MirroredTexture;


#ifdef __HIPCC__

/**
 * https://stackoverflow.com/questions/10535667/does-it-make-any-sense-to-use-inline-keyword-with-templates
 */
template< class T >
class MirroredVector
{
    #define DEBUG_MIRRORED_VECTOR 0
public:
    typedef T value_type;

    T *                host     ;
    T *                gpu      ;
    size_t       const nElements;
    size_t       const nBytes   ;
    hipStream_t const mStream  ;
    bool         const mAsync   ;

    inline MirroredVector()
     : host( NULL ), gpu( NULL ), nElements( 0 ), nBytes( 0 ), mStream( 0 ), mAsync( false )
    {}

    inline void malloc()
    {
        if ( host == NULL )
        {
            #if DEBUG_MIRRORED_VECTOR > 10
                std::cerr << "[" << __FILENAME__ << "::MirroredVector::malloc]"
                    << "Allocate " << prettyPrintBytes( nBytes ) << " on host.\n";
            #endif
            host = (T*) ::malloc( nBytes );
        }
        if ( gpu == NULL )
        {
            #if DEBUG_MIRRORED_VECTOR > 10
                std::cerr << "[" << __FILENAME__ << "::MirroredVector::malloc]"
                    << "Allocate " << prettyPrintBytes( nBytes ) << " on GPU.\n";
            #endif
            CUDA_ERROR( hipMalloc( (void**) &gpu, nBytes ) );
        }
        if ( ! ( host != NULL && gpu != NULL ) )
        {
            std::stringstream msg;
            msg << "[" << __FILENAME__ << "::MirroredVector::malloc] "
                << "Something went wrong when trying to allocate memory "
                << "(host=" << (void*) host << ", gpu=" << (void*) gpu
                << ", nBytes=" << nBytes << std::endl;
            throw std::runtime_error( msg.str() );
        }
    }

    inline MirroredVector
    (
        size_t const rnElements,
        hipStream_t rStream = 0,
        bool const   rAsync  = false
    )
     : host( NULL ), gpu( NULL ), nElements( rnElements ),
       nBytes( rnElements * sizeof(T) ), mStream( rStream ),
       mAsync( rAsync )
    {
        this->malloc();
    }

    /**
     * Uses async, but not that by default the memcpy gets queued into the
     * same stream as subsequent kernel calls will, so that a synchronization
     * will be implied
     * @param[in] rAsync -1 uses the default as configured using the constructor
     *                    0 (false) synchronizes stream after memcpyAsync
     *                    1 (true ) will transfer asynchronously
     */
    inline void push( int const rAsync = -1 ) const
    {
        if ( ! ( host != NULL || gpu != NULL || nBytes == 0 ) )
        {
            std::stringstream msg;
            msg << "[" << __FILENAME__ << "::MirroredVector::push] "
                << "Can't push, need non NULL pointers and more than 0 elements. "
                << "(host=" << (void*) host << ", gpu=" << (void*) gpu
                << ", nBytes=" << nBytes << std::endl;
            throw std::runtime_error( msg.str() );
        }
        CUDA_ERROR( hipMemcpyAsync( (void*) gpu, (void*) host, nBytes,
                                     hipMemcpyHostToDevice, mStream ) );
        CUDA_ERROR( hipPeekAtLastError() );
        if ( ( rAsync == -1 && ! mAsync ) || ! rAsync )
            CUDA_ERROR( hipStreamSynchronize( mStream ) );
    }
    inline void pushAsync( void ) const { push( true ); }

    inline void pop( int const rAsync = -1 ) const
    {
        if ( ! ( host != NULL || gpu != NULL || nBytes == 0 ) )
        {
            std::stringstream msg;
            msg << "[" << __FILENAME__ << "::MirroredVector::pop] "
                << "Can't pop, need non NULL pointers and more than 0 elements. "
                << "(host=" << (void*) host << ", gpu=" << (void*) gpu
                << ", nBytes=" << nBytes << std::endl;
            throw std::runtime_error( msg.str() );
        }
        CUDA_ERROR( hipMemcpyAsync( (void*) host, (void*) gpu, nBytes,
                                     hipMemcpyDeviceToHost, mStream ) );
        CUDA_ERROR( hipPeekAtLastError() );
        if ( ( rAsync == -1 && ! mAsync ) || ! rAsync )
            CUDA_ERROR( hipStreamSynchronize( mStream ) );
    }
    inline void popAsync( void ) const { pop( true ); }

    inline void free()
    {
        if ( host != NULL )
        {
            ::free( host );
            host = NULL;
        }
        if ( gpu != NULL )
        {
            CUDA_ERROR( hipFree( gpu ) );
            gpu = NULL;
        }
    }

    inline ~MirroredVector()
    {
        this->free();
    }

    #undef DEBUG_MIRRORED_VECTOR
};

template< typename T >
std::ostream & operator<<( std::ostream & out, MirroredVector<T> const & x )
{
    out << "( nElements = " << x.nElements << ", "
        << "nBytes = " << x.nBytes << ","
        << "sizeof(T) = " << sizeof(T) << ","
        << "host = " << x.host << ","
        << "gpu = " << x.gpu << " )";
    return out;
}

template< class T >
class MirroredTexture : public MirroredVector<T>
{
public:
    hipResourceDesc    mResDesc;
    hipTextureDesc     mTexDesc;
    hipTextureObject_t texture ;

    /**
     * @see http://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TEXTURE__OBJECT.html
     * @see https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/
     * @see http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-memory
     */
    inline void bind()
    {
        memset( &mResDesc, 0, sizeof( mResDesc ) );
        /**
         * enum hipResourceType
         *   hipResourceTypeArray          = 0x00
         *   hipResourceTypeMipmappedArray = 0x01
         *   hipResourceTypeLinear         = 0x02
         *   hipResourceTypePitch2D        = 0x03
         */
        mResDesc.resType = hipResourceTypeLinear;
        /**
         * enum hipChannelFormatKind
         *   hipChannelFormatKindSigned   = 0
         *   hipChannelFormatKindUnsigned = 1
         *   hipChannelFormatKindFloat    = 2
         *   hipChannelFormatKindNone     = 3
         */
        mResDesc.res.linear.desc.f      = hipChannelFormatKindUnsigned;
        mResDesc.res.linear.desc.x      = sizeof(T) * 8; // bits per channel
        mResDesc.res.linear.devPtr      = this->gpu;
        mResDesc.res.linear.sizeInBytes = this->nBytes;

        memset( &mTexDesc, 0, sizeof( mTexDesc ) );
        /**
         * enum hipTextureReadMode
         *   hipReadModeElementType     = 0
         *     Read texture as specified element type
         *   hipReadModeNormalizedFloat = 1
         *     Read texture as normalized float
         */
        mTexDesc.readMode = hipReadModeElementType;

        /* the last three arguments are pointers to constants! */
        hipCreateTextureObject( &texture, &mResDesc, &mTexDesc, NULL );
    }

    inline MirroredTexture
    (
        size_t const rnElements,
        hipStream_t rStream = 0,
        bool const   rAsync  = false
    )
     : MirroredVector<T>( rnElements, rStream, rAsync ), texture( 0 )
    {
        this->bind();
    }

    inline ~MirroredTexture()
    {
        hipDestroyTextureObject( texture );
        texture = 0;
        this->free();
    }
};

#endif // __HIPCC__


template< class T >
inline __device__ __host__
void swap( T & a, T & b )
{
    T const c = a;
    a = b;
    b = c;
}


template< typename T >
inline __device__ __host__
int snprintInt
(
    char             * const msg  ,
    unsigned int       const nChars,
    T                        number,
    unsigned short int const base = 10u
)
{
    assert( base <= ( '9' - '0' + 1 ) + ( 'Z' - 'A' + 1 ) && "base was chosen too high, not sure how to convert that to characters!" );

    unsigned int nCharsWritten = 0u;
    if ( nCharsWritten+1 >= nChars )
        return 0;
    else if ( number < 0 )
    {
        msg[ nCharsWritten++ ] = '-';
        number = -number;
    }

    unsigned int expFloorLogBase = 1;
    while ( number / expFloorLogBase >= base )
        expFloorLogBase *= base;

    /* e.g. a possible run for 1230:
     *   digit 0 = 1 = 1230 / 1000
     *   digit 1 = 2 = 230  / 100
     *   digit 2 = 3 = 30   / 10
     *   digit 3 = 0 = 0    / 1 */
    while ( expFloorLogBase != 0 )
    {
        unsigned int const digit = number / expFloorLogBase;
        number          %= expFloorLogBase;
        expFloorLogBase /= base;
        assert( digit <= base );

        if ( nCharsWritten+1 < nChars )
        {
            if ( digit < '9' - '0' + 1 )
                msg[ nCharsWritten++ ] = '0' + (unsigned char) digit;
            else if ( digit - ( '9' - '0' + 1 ) < 'Z' - 'A' + 1u )
                msg[ nCharsWritten++ ] = 'Z' + (unsigned char)( digit - ( '9' - '0' + 1u ) );
            else
                assert( false && "base was chosen too high, not sure how to convert that to characters!" );
        }
        else
            break;
    }

    assert( nCharsWritten+1 <= nChars ); // includes nChars > 0
    msg[ nCharsWritten ] = '\0';
    return nCharsWritten;
}

inline __device__ __host__
int snprintFloatArray
(
    char        * const msg  ,
    unsigned int  const nChars,
    float const * const gpData,
    unsigned int  const nElements
)
{
    unsigned int nCharsWritten = 0u;
    for ( unsigned int j = 0u; j < nElements; ++j )
    {
        if ( nCharsWritten + 1 >= nChars )
            break;
        msg[ nCharsWritten++ ] = ' ';
        //nCharsWritten += snprintFloat( msg, nChars - nCharsWritten, gpData[j] );
        nCharsWritten += snprintInt( msg + nCharsWritten, nChars - nCharsWritten, (int)( 10000 * gpData[j] ) );
    }
    assert( nCharsWritten < nChars );
    msg[ nCharsWritten ] = '\0';
    return nCharsWritten;
}

#if __cplusplus >= 201103

/**
 * @see https://stackoverflow.com/questions/18625964/checking-if-an-input-is-within-its-range-of-limits-in-c
 * Use e.g. like this:
 *   int32_t value = 123456;
 *   assert( inRange< uint16_t >( value ) ); // will fail, because max. is 65535
 */
#include <cmath>                            // isnan, isinf
#include <limits>
#include <type_traits>                      // remove_reference

template< typename T_Range, typename T_Value, bool T_RangeSigned, bool T_ValueSigned >
struct InIntegerRange;

template< typename T_Range, typename T_Value >
struct InIntegerRange< T_Range, T_Value, false, false >
{
    bool operator()( T_Value const & x )
    {
        return x >= std::numeric_limits< T_Range >::min() &&
               x <= std::numeric_limits< T_Range >::max();
    }
};

template< typename T_Range, typename T_Value >
struct InIntegerRange< T_Range, T_Value, false, true >
{
    bool operator()( T_Value const & x )
    {
        return x >= 0 && x <= std::numeric_limits< T_Range >::max();
    }
};

template< typename T_Range, typename T_Value >
struct InIntegerRange< T_Range, T_Value, true, false >
{
    bool operator()( T_Value const & x )
    {
        return x <= std::numeric_limits< T_Range >::max(); /* x >= 0 is given */
    }
};

template< typename T_Range, typename T_Value >
struct InIntegerRange< T_Range, T_Value, true, true >
{
    bool operator()( T_Value const & x )
    {
        return x >= std::numeric_limits< T_Range >::min() &&
               x <= std::numeric_limits< T_Range >::max();
    }
};

template< typename T_Range, typename T_Value >
inline bool inRange( T_Value const & x )
{
    using Range = typename std::remove_reference< T_Range >::type;
    using Value = typename std::remove_reference< T_Value >::type;

    if( std::numeric_limits< Range >::is_integer )
    {
        return InIntegerRange< Range, Value,
                               std::numeric_limits< Range >::is_signed,
                               std::numeric_limits< Value >::is_signed >()( x );
    }
    else
    {
        return ( x > 0 ? x : -x ) <= std::numeric_limits< Range >::max() ||
               ( std::isnan(x) && std::numeric_limits< Range >::has_quiet_NaN ) ||
               ( std::isinf(x) && std::numeric_limits< Range >::has_infinity );
    }
}

#endif

#ifdef CUDACOMMON_GPUINFO_MAIN
int main( void )
{
    hipDeviceProp_t * pGpus = NULL;
    int              nGpus = 0   ;
    getCudaDeviceProperties( &pGpus, &nGpus, true );
    return 0;
}
#endif
