#include "hip/hip_runtime.h"
#pragma once

#include <cassert>
#include <cstdint>
#include <cstdio>
#include <cstdlib>    // EXIT_FAILURE, exit
#include <iostream>


/**
 * compile with:
 *   'nvcc' -ccbin=/usr/bin/g++-4.9 -std=c++11 --compiler-options -Wall,-Wextra -DCUDACOMMON_GPUINFO_MAIN -o gpuinfo --x cu gpuinfo.cu
 */


inline void checkCudaError(const hipError_t rValue, const char * file, int line )
{
    if ( (rValue) != hipSuccess )
    {
        std::cout << "CUDA error in " << file
                  << " line:" << line << " : "
                  << hipGetErrorString(rValue) << "\n";
        assert( false );
    }
}
#define CUDA_ERROR(X) checkCudaError( X, __FILE__, __LINE__ );


template< typename T, typename S >
__host__ __device__
inline T ceilDiv( T a, S b )
{
    assert( b != 0 );
    assert( a == a );
    assert( b == b );
    return (a+b-1)/b;
}

/**
 * Chooses an optimal configuration for number of blocks and number of threads
 * Note that every kernel may have to calculate on a different amount of
 * elements, so this needs to be calculated inside the kernel with:
 *    for ( i = linid; i < nElements; i += nBlocks * nThreads )
 * which yields the following number of iterations:
 *    nIterations = (nElements-1 - linid) / ( nBlocks * nThreads ) + 1
 * derivation:
 *    search for highest n which satisfies i + n*s <= m-1
 *    note that we used <= m-1 instead of < m to work with floor later on
 *    <=> search highest n: n <= (m-1-i)/s
 *    which is n = floor[ (m-1-i)/s ]. Note that floor wouldn't be possible
 *    for < m, because it wouldn't account for the edge case for (m-1-i)/s == n
 *    the highest n means the for loop will iterate with i, i+s, i+2*s, i+...n*s
 *    => nIterations = n+1 = floor[ (m-1-i)/s ] + 1
 */
inline void calcKernelConfig( int iDevice, uint64_t n, int * nBlocks, int * nThreads )
{
    int const nMaxThreads  = 256;
    int const nMinElements = 32; /* The assumption: one kernel with nMinElements work won't be much slower than nMinElements kernels with each 1 work element. Of course this is workload / kernel dependent, so the fixed value may not be the best idea */

    /* set current device and get device infos */
    int nDevices;
    CUDA_ERROR( hipGetDeviceCount( &nDevices ) );
    assert( iDevice < nDevices );
    CUDA_ERROR( hipSetDevice( iDevice ) );

    // for GTX 760 this is 12288 threads per device and 384 real cores
    hipDeviceProp_t deviceProperties;
    CUDA_ERROR( hipGetDeviceProperties( &deviceProperties, iDevice) );

    int const nMaxThreadsGpu = deviceProperties.maxThreadsPerMultiProcessor
                             * deviceProperties.multiProcessorCount;
    if ( n < (uint64_t) nMaxThreadsGpu * nMinElements )
    {
        auto nThreadsNeeded = ceilDiv( n, nMinElements );
        *nBlocks  = ceilDiv( nThreadsNeeded, nMaxThreads );
        *nThreads = nMaxThreads;
        if ( *nBlocks == 1 )
        {
            assert( nThreadsNeeded <= nMaxThreads );
            *nThreads = nThreadsNeeded;
        }
    }
    else
    {
        *nBlocks  = nMaxThreadsGpu / nMaxThreads;
        *nThreads = nMaxThreads;
    }
    assert( *nBlocks > 0 );
    assert( *nThreads > 0 );
    uint64_t nIterations = 0;
    for ( uint64_t linid = 0; linid < (uint64_t) *nBlocks * *nThreads; ++linid )
    {
        /* note that this only works if linid < n */
        assert( linid < n );
        nIterations += (n-linid-1) / ( *nBlocks * *nThreads ) + 1;
        //printf( "[thread %i] %i elements\n", linid, (n-linid) / ( *nBlocks * *nThreads ) );
    }
    //printf( "Total %i elements out of %i wanted\n", nIterations, n );
    assert( nIterations == n );
}


template< class T >
struct GpuArray
{
    T * host, * gpu;
    unsigned long long int const nBytes;
    hipStream_t mStream;

    inline GpuArray
    (
        unsigned long long int const nElements = 1,
        hipStream_t rStream = 0
    )
    : nBytes( nElements * sizeof(T) ),
      mStream( rStream )
    {
        host = (T*) malloc( nBytes );
        CUDA_ERROR( hipMalloc( (void**) &gpu, nBytes ) );
        assert( host != NULL );
        assert( gpu  != NULL );
    }
    inline ~GpuArray()
    {
        CUDA_ERROR( hipFree( gpu ) );
        free( host );
    }
    inline void down( void )
    {
        CUDA_ERROR( hipMemcpyAsync( (void*) host, (void*) gpu, nBytes, hipMemcpyDeviceToHost ) );
        CUDA_ERROR( hipPeekAtLastError() );
    }
    inline void up( void )
    {
        CUDA_ERROR( hipMemcpyAsync( (void*) gpu, (void*) host, nBytes, hipMemcpyHostToDevice ) );
        CUDA_ERROR( hipPeekAtLastError() );
    }
};


inline
__device__
long long unsigned int getLinearThreadId( void )
{
    long long unsigned int i    = threadIdx.x;
    long long unsigned int iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax;
    // expands to: i = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x

    return i;
}

inline
__device__
long long unsigned int getLinearGlobalId( void )
{
    long long unsigned int i    = threadIdx.x;
    long long unsigned int iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax; iMax *= blockDim.z;
    i +=  blockIdx.x * iMax; iMax *= gridDim.x;
    i +=  blockIdx.y * iMax; iMax *= gridDim.y;
    i +=  blockIdx.z * iMax;

    return i;
}

#include <utility>                  // pair

inline
__device__
void getLinearGlobalIdSize
(
    long long unsigned int * riThread,
    long long unsigned int * rnThreads
)
{
    auto & i    = *riThread ;
    auto & iMax = *rnThreads;

    i    = threadIdx.x;
    iMax = blockDim.x;

    i += threadIdx.y * iMax; iMax *= blockDim.y;
    i += threadIdx.z * iMax; iMax *= blockDim.z;
    i +=  blockIdx.x * iMax; iMax *= gridDim.x;
    i +=  blockIdx.y * iMax; iMax *= gridDim.y;
    i +=  blockIdx.z * iMax; iMax *= gridDim.z;
}

inline
__device__
long long unsigned int getLinearBlockId( void )
{
    long long unsigned int i    = blockIdx.x;
    long long unsigned int iMax = gridDim.x;

    i += blockIdx.y * iMax; iMax *= gridDim.y;
    i += blockIdx.z * iMax;
    // expands to: i = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x

    return i;
}

inline
__device__
long long unsigned int getBlockSize( void )
{
    return blockDim.x * blockDim.y * blockDim.z;
}

inline
__device__
long long unsigned int getGridSize( void )
{
    return gridDim.x * gridDim.y * gridDim.z;
}


#include <cassert>
#include <cstdio>               // printf, fflush
#include <cstdlib>              // NULL, malloc, free


/**
 * Returns the number of arithmetic CUDA cores per streaming multiprocessor
 * Note that there are also extra special function units.
 * Note that for 2.0 the two warp schedulers can only issue 16 instructions
 * per cycle each. Meaning the 32 CUDA cores can't be used in parallel with
 * the 4 special function units. For 2.1 up this is a different matter
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
 **/
int getCudaCoresPerMultiprocessor
(
    int const majorVersion,
    int const minorVersion
)
{
    if ( majorVersion == 2 && minorVersion == 0 ) /* Fermi */
        return 32;
    if ( majorVersion == 2 && minorVersion == 1 ) /* Fermi */
        return 48;
    if ( majorVersion == 3 )  /* Kepler */
        return 192;
    if ( majorVersion == 5 )  /* Maxwell */
        return 128;
    if ( majorVersion == 6 )  /* Pascal */
        return 64;
    return 0;   /* unknown, could also throw exception= */
}

std::string getCudaCodeName
(
    int const majorVersion,
    int const = 0
)
{
    if ( majorVersion == 2 )
        return "Fermi";
    if ( majorVersion == 3 )
        return "Kepler";
    if ( majorVersion == 5 )
        return "Maxwell";
    if ( majorVersion == 6 )
        return "Pascal";
    return 0;   /* unknown, could also throw exception= */
}

/**
 * @return flops (not GFlops, ... )
 */
float getCudaPeakFlops( hipDeviceProp_t const & props )
{
    return (float) props.multiProcessorCount * props.clockRate /* kHz */ * 1e3f *
        getCudaCoresPerMultiprocessor( props.major, props.minor );
}

/**
 * @param[out] rpDeviceProperties - Array of hipDeviceProp_t of length rnDevices
 *             the user will need to free (C-style) this data on program exit!
 * @param[out] rnDevices - will hold number of cuda devices
 **/
inline void getCudaDeviceProperties
(
    hipDeviceProp_t **       rpDeviceProperties = NULL,
    int             *       rnDevices          = NULL,
    bool              const rPrintInfo         = true
)
{
    printf( "Getting Device Informations. As this is the first command, "
            "it can take ca.30s, because the GPU must be initialized.\n" );
    fflush( stdout );

    int fallbackNDevices;
    if ( rnDevices == NULL )
        rnDevices = &fallbackNDevices;
    CUDA_ERROR( hipGetDeviceCount( rnDevices ) );

    hipDeviceProp_t * fallbackPropArray;
    if ( rpDeviceProperties == NULL )
        rpDeviceProperties = &fallbackPropArray;
    *rpDeviceProperties = (hipDeviceProp_t*) malloc( (*rnDevices) * sizeof(hipDeviceProp_t) );
    assert( *rpDeviceProperties != NULL );

    for ( int iDevice = 0; iDevice < (*rnDevices); ++iDevice )
    {
        hipDeviceProp_t * prop = &( (*rpDeviceProperties)[iDevice] );
        CUDA_ERROR( hipGetDeviceProperties( prop, iDevice ) );

		if ( not rPrintInfo )
			continue;

        if ( iDevice == 0 && prop->major == 9999 && prop->minor == 9999 )
            printf("There is no device supporting CUDA.\n");

		const char cms[5][20] =
			{ "Default", "Exclusive", "Prohibited", "ExclusiveProcess", "Unknown" };
		const char * computeModeString;
		switch ( prop->computeMode )
        {
			case hipComputeModeDefault          : computeModeString = cms[0];
			case hipComputeModeExclusive        : computeModeString = cms[1];
			case hipComputeModeProhibited       : computeModeString = cms[2];
			case hipComputeModeExclusiveProcess : computeModeString = cms[3];
			default                              : computeModeString = cms[4];
		}
        auto const coresPerSM = getCudaCoresPerMultiprocessor( prop->major, prop->minor );
        auto const peakFlops  = getCudaPeakFlops( *prop );

        printf( "\n================== Device Number %i ==================\n",iDevice );
        printf( "| Device name              : %s\n"        , prop->name );
        printf( "| Computability            : %i.%i\n"     , prop->major,
                                                             prop->minor );
        printf( "| Code Name                : %s\n"        , getCudaCodeName( prop->major, prop->minor ).c_str() );
        printf( "| PCI Bus ID               : %i\n"        , prop->pciBusID );
        printf( "| PCI Device ID            : %i\n"        , prop->pciDeviceID );
        printf( "| PCI Domain ID            : %i\n"        , prop->pciDomainID );
		printf( "|------------------- Architecture -------------------\n" );
        printf( "| Number of SMX            : %i\n"        , prop->multiProcessorCount );
        printf( "| Max Threads per SMX      : %i\n"        , prop->maxThreadsPerMultiProcessor );
        printf( "| Max Threads per Block    : %i\n"        , prop->maxThreadsPerBlock );
        printf( "| Warp Size                : %i\n"        , prop->warpSize );
        printf( "| Clock Rate               : %f GHz\n"    , prop->clockRate/1.0e6f );
        printf( "| Max Block Size           : (%i,%i,%i)\n", prop->maxThreadsDim[0],
                                                             prop->maxThreadsDim[1],
                                                             prop->maxThreadsDim[2] );
        printf( "| Max Grid Size            : (%i,%i,%i)\n", prop->maxGridSize[0],
                                                             prop->maxGridSize[1],
                                                             prop->maxGridSize[2] );
		printf( "|  => Max conc. Threads    : %i\n"        , prop->multiProcessorCount *
		                                                     prop->maxThreadsPerMultiProcessor );
		printf( "|  => Warps per SMX        : %i\n"        , prop->maxThreadsPerMultiProcessor /
		                                                     prop->warpSize );
        printf( "| CUDA Cores per Multiproc.: %i\n"        , coresPerSM );
        printf( "| Total CUDA Cores         : %i\n"        , prop->multiProcessorCount * coresPerSM );
        printf( "| Clock Rate               : %f GHz\n"    , peakFlops / 1e9f );
		printf( "|---------------------- Memory ----------------------\n" );
        printf( "| Total Global Memory      : %lu Bytes\n" , prop->totalGlobalMem );
        printf( "| Total Constant Memory    : %lu Bytes\n" , prop->totalConstMem );
        printf( "| Shared Memory per Block  : %lu Bytes\n" , prop->sharedMemPerBlock );
        printf( "| L2 Cache Size            : %u Bytes\n"  , prop->l2CacheSize );
        printf( "| Registers per Block      : %i\n"        , prop->regsPerBlock );
        printf( "| Memory Bus Width         : %i Bits\n"   , prop->memoryBusWidth );
        printf( "| Memory Clock Rate        : %f GHz\n"    , prop->memoryClockRate/1.0e6f );
        printf( "| Memory Pitch             : %lu\n"       , prop->memPitch );
        printf( "| Unified Addressing       : %i\n"        , prop->unifiedAddressing );
        printf( "| Texture Alignment        :  %ld\n"      , prop->textureAlignment );
        printf( "| Max 1D Texture Size      : %i\n"        , prop->maxTexture1D );
        printf( "| Max 2D Texture Size      : (%i,%i)\n"   , prop->maxTexture2D[0],
                                                             prop->maxTexture2D[1] );
        // this really is ONLY in CUDA 3.2. (maybe 3.x) available Oo
        //printf( "| Max 2D Texture Array Size: (%i,%i)\n"   , prop->maxTexture2DArray[0],
        //                                                     prop->maxTexture2DArray[1] );
        printf( "| Max 3D Texture Size      : (%i,%i,%i)\n", prop->maxTexture3D[0],
                                                             prop->maxTexture3D[1] ,
                                                             prop->maxTexture3D[2] );
		printf( "|--------------------- Graphics ---------------------\n" );
		printf( "| Compute mode             : %s\n"        ,      computeModeString );
		printf( "|---------------------- Other -----------------------\n" );
        printf( "| Can map Host Memory      : %s\n"        , prop->canMapHostMemory  ? "true" : "false" );
        printf( "| Can run Kernels conc.    : %s\n"        , prop->concurrentKernels ? "true" : "false" );
		printf( "| Number of Asyn. Engines  : %i\n"        , prop->asyncEngineCount );
        printf( "| Can Copy and Kernel conc.: %s\n"        , prop->deviceOverlap     ? "true" : "false" );
        printf( "| ECC Enabled              : %s\n"        , prop->ECCEnabled        ? "true" : "false" );
        printf( "| Device is Integrated     : %s\n"        , prop->integrated        ? "true" : "false" );
        printf( "| Kernel Timeout Enabled   : %s\n"        , prop->kernelExecTimeoutEnabled ? "true" : "false" );
        printf( "| Uses TESLA Driver        : %s\n"        , prop->tccDriver         ? "true" : "false" );
        printf( "=====================================================\n" );
    }
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
/**
 * atomicAdd for double is not natively implemented, because it's not
 * supported by (all) the hardware, therefore resulting in a time penalty.
 * http://stackoverflow.com/questions/12626096/why-has-atomicadd-not-been-implemented-for-doubles
 */
inline __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int *) address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
              __double_as_longlong( val + __longlong_as_double(assumed) ));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif


template< class T >
class MirroredVector
{
public:
    T * cpu = NULL;
    T * gpu = NULL;
    size_t n;

    MirroredVector() : n( 0 ), cpu( NULL ), gpu( NULL ) {}
    MirroredVector( size_t const rN ) : n( rN )
    {
        cpu = new T[ n ];
        CUDA_ERROR( hipMalloc( (void**) &gpu, n * sizeof(T) ) );
        assert( cpu != NULL );
        assert( gpu != NULL );
    }
    void push( void ) const
    {
        assert( cpu != NULL );
        assert( gpu != NULL );
        assert( n > 0 );
        CUDA_ERROR( hipMemcpy( (void*) gpu, (void*) cpu, n * sizeof(T), hipMemcpyHostToDevice ) );
        CUDA_ERROR( hipPeekAtLastError() );
    }
    void pop( void ) const
    {
        assert( cpu != NULL );
        assert( gpu != NULL );
        assert( n > 0 );
        CUDA_ERROR( hipMemcpyAsync( (void*) cpu, (void*) gpu, n * sizeof(T), hipMemcpyDeviceToHost ) );
        CUDA_ERROR( hipPeekAtLastError() );
    }
    ~MirroredVector()
    {
        if ( cpu != NULL )
        {
            delete[] cpu;
            cpu = NULL;
        }
        if ( gpu != NULL )
        {
            CUDA_ERROR( hipFree( gpu ) );
            gpu = NULL;
        }
    }
};


template< class T >
__device__ void swap( T & a, T & b )
{
    auto const c = a;
    a = b;
    b = c;
}


__device__
inline
int snprintFloat
(
    char        * const msg  ,
    unsigned int  const nChars,
    float         const f
)
{
    assert( false && "unfinished skeleton" );
    return 0;
}

template< typename T >
__device__
inline
int snprintInt
(
    char             * const msg  ,
    unsigned int       const nChars,
    T                        number,
    unsigned short int const base = 10u
)
{
    assert( base <= ( '9' - '0' + 1 ) + ( 'Z' - 'A' + 1 ) && "base was chosen too high, not sure how to convert that to characters!" );

    unsigned int nCharsWritten = 0u;
    if ( nCharsWritten+1 >= nChars )
        return 0;
    else if ( number < 0 )
    {
        msg[ nCharsWritten++ ] = '-';
        number = -number;
    }

    unsigned int expFloorLogBase = 1;
    while ( number / expFloorLogBase >= base )
        expFloorLogBase *= base;

    /* e.g. a possible run for 1230:
     *   digit 0 = 1 = 1230 / 1000
     *   digit 1 = 2 = 230  / 100
     *   digit 2 = 3 = 30   / 10
     *   digit 3 = 0 = 0    / 1 */
    while ( expFloorLogBase != 0 )
    {
        auto const digit = number / expFloorLogBase;
        number          %= expFloorLogBase;
        expFloorLogBase /= base;
        assert( digit <= base );

        if ( nCharsWritten+1 < nChars )
        {
            if ( digit < '9' - '0' + 1 )
                msg[ nCharsWritten++ ] = '0' + (unsigned char) digit;
            else if ( digit - ( '9' - '0' + 1 ) < 'Z' - 'A' + 1u )
                msg[ nCharsWritten++ ] = 'Z' + (unsigned char)( digit - ( '9' - '0' + 1u ) );
            else
                assert( false && "base was chosen too high, not sure how to convert that to characters!" );
        }
        else
            break;
    }

    assert( nCharsWritten+1 <= nChars ); // includes nChars > 0
    msg[ nCharsWritten ] = '\0';
    return nCharsWritten;
}

__device__
inline
int snprintFloatArray
(
    char        * const msg  ,
    unsigned int  const nChars,
    float const * const gpData,
    unsigned int  const nElements
)
{
    unsigned int nCharsWritten = 0u;
    for ( auto j = 0u; j < nElements; ++j )
    {
        if ( nCharsWritten + 1 >= nChars )
            break;
        msg[ nCharsWritten++ ] = ' ';
        //nCharsWritten += snprintFloat( msg, nChars - nCharsWritten, gpData[j] );
        nCharsWritten += snprintInt( msg + nCharsWritten, nChars - nCharsWritten, (int)( 10000 * gpData[j] ) );
    }
    assert( nCharsWritten < nChars );
    msg[ nCharsWritten ] = '\0';
    return nCharsWritten;
}

#ifdef CUDACOMMON_GPUINFO_MAIN
int main( void )
{
    hipDeviceProp_t * pGpus = NULL;
    int              nGpus = 0   ;
    getCudaDeviceProperties( &pGpus, &nGpus, true );
    return 0;
}
#endif
